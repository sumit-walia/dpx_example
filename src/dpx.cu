#include "hip/hip_runtime.h"
#include <iostream>
#include "dpx.cuh"
#include <stdio.h>

void printGpuProperties () {
    int nDevices;

    // Store the number of available GPU device in nDevicess
    hipError_t err = hipGetDeviceCount(&nDevices);

    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipGetDeviceCount failed!\n");
        exit(1);
    }

    // For each GPU device found, print the information (memory, bandwidth etc.)
    // about the device
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Device memory: %lu\n", prop.totalGlobalMem);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}

__global__ void smith_waterman_call() {

    int match = 2;
    int mismatch = -1;
    int gap = -1;

    char ref[] = "ACGTAAC";
    char query[] = "ACTATC";

    int rlen = 7;
    int qlen = 6;

    int score_matrix[8][7] = {};

    score_matrix[0][0] = 0;

    for (int i = 1; i < rlen + 1; i++)
    {
        score_matrix[i][0] = score_matrix[i-1][0] + gap;
    }

    for (int i = 1; i < qlen + 1; i++)
    {
        score_matrix[0][i] = score_matrix[0][i-1] + gap;
    }

    for (int i = 1; i < rlen + 1; i++)
    {
        for (int j = 1; j < qlen + 1; j++)
        {
            int up = score_matrix[i-1][j] + gap;
            int left = score_matrix[i][j-1]  + gap;
            int diag = score_matrix[i-1][j-1];
            if (ref[i-1] == query[j-1])
                diag += match;
            else
                diag += mismatch;

            score_matrix[i][j] = __vimax3_s32(diag, left, up); 
        }
    }

    printf("\t");
    for (int j = 0; j < qlen; j++)
    {
        printf("%c\t", query[j]); 
    }
    printf("\n"); 

    for (int i = 1; i < rlen + 1; i++)
    {
        printf("%c\t", ref[i-1]); 
        for (int j = 1; j < qlen + 1; j++)
        {
            printf("%d\t", score_matrix[i][j]); 
        }
        printf("\n"); 
    }

}

void dpx::smith_waterman() {

    int numBlocks = 1; // i.e. number of thread blocks on the GPU
    int blockSize = 1; // i.e. number of GPU threads per thread block

    smith_waterman_call<<<numBlocks, blockSize>>>();

    hipDeviceSynchronize();
}